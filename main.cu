#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <fstream>
#include <cstdlib>
#include <chrono>
#include <Eigen/Dense>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <omp.h>

#define NUM_THREADS 6;

int main(int argc, char* argv[])
{
    std::cout<<"\n***** Starting Cholesky Solvers *****"<<std::endl;

    int sizes[] = {200, 500, 800, 1000, 2000, 3000, 4000, 5000};
    Eigen::setNbThreads(6); // To enable multithreaded computation, if and when available.
    std::chrono::steady_clock::time_point start;
    std::chrono::steady_clock::time_point stop;

    std::ofstream out("measurements.csv"); // to store the measurement data
    out<<"Resolution,CPU,GPU,GPU-Speedup\n";

    for(int j = 0; j < 8; j++)
    {
        int size = sizes[j];        

        // Create CUDA instances and handles
        hipError_t cudaStatus;
        hipsolverStatus_t cusolverStatus ;
        hipsolverHandle_t handle;
        cusolverStatus = hipsolverDnCreate(&handle);

        /* Create data structures in Eigen. 
           Equation considered is Ax=b, with A = I, b and x are random
           In the end, the solution should be x = b.
        */
        
        Eigen::MatrixXf A(size, size);
        Eigen::VectorXf b(size);
        Eigen::VectorXf x(size);
        A = Eigen::MatrixXf::Identity(size, size);
        b = Eigen::VectorXf::Random(size);
        x = Eigen::VectorXf::Random(size);

        // Obtain pointers of the Eigen data so that they can be copied into Thrust vectors
        float *A_eigen = A.data(); 
        float *b_eigen = b.data();
        int Lwork, *d_info;

        // Create host and device data in Thrust
        thrust::host_vector<float> mat_A(A_eigen, A_eigen + A.size());
        thrust::host_vector<float> vec_b(b_eigen, b_eigen + b.size());
        thrust::host_vector<float> info;
        thrust::device_vector<float> d_A = mat_A;
        thrust::device_vector<float> d_b = vec_b;    
        cudaStatus = hipMalloc((void **) &d_info, sizeof(int));  

        hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER ;
        cusolverStatus = hipsolverDnSpotrf_bufferSize(handle, uplo, size, d_A.data().get(), size, &Lwork);
        
        thrust::host_vector<float> Work(Lwork);
        thrust::device_vector<float> d_Work = Work;

        // Solve on the GPU
        start = std::chrono::steady_clock::now();

        cusolverStatus = hipsolverDnSpotrf(handle, uplo, size, d_A.data().get(), size, d_Work.data().get(), Lwork, d_info);
        cusolverStatus = hipsolverDnSpotrs(handle, uplo, size, 1, d_A.data().get(), size, d_b.data().get(), size, d_info);
        cudaStatus = hipDeviceSynchronize();
        
        stop = std::chrono::steady_clock::now();
        auto duration_gpu = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();

        // Solve on the CPU
        start = std::chrono::steady_clock::now();

        Eigen::LDLT<Eigen::MatrixXf> ldlt(size);
        ldlt.compute(A);
        x = ldlt.solve(b);

        stop = std::chrono::steady_clock::now();
        auto duration_cpu = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
        auto speedup = duration_cpu / duration_gpu;

        // Check results
        vec_b = d_b; // get the device vector pointers
        Eigen::Map<Eigen::VectorXf> x_gpu(vec_b.data(), x.size()); // map the device vector to a host container  

        std::cout<<"\nResolution : "<<size<<std::endl;
        std::cout<<"\tGPU (milliseconds) : "<<duration_gpu / 1000;
        std::cout<<"\n\tCPU (milliseconds) : "<<duration_cpu / 1000;
        std::cout<<"\n\tDifference (in 2 Norm) : "<<(x - x_gpu).squaredNorm()<<std::endl;
        std::cout<<"\n\tGPU Speedup : "<<speedup<<std::endl;

        // Write measurements to file
        out<<size<<","<<duration_cpu / 1000<<","<<duration_gpu / 1000<<","<<speedup<<"\n";
    }

    out.close();    
    return 0;
}